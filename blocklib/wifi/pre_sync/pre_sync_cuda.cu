#include "hip/hip_runtime.h"
/* -*- c++ -*- */
/*
 * Copyright 2022 Josh Morman
 *
 * This file is part of GNU Radio
 *
 * SPDX-License-Identifier: GPL-3.0-or-later
 *
 */

#include "pre_sync_cuda.cuh"

// The block cuda file is just a wrapper for the kernels that will be launched in the work
// function
namespace gr {
namespace wifi {
namespace pre_sync_cu {

__global__ void
corr_abs_kernel(hipFloatComplex* in, hipFloatComplex* out, float* mag, int n)
{
    int d = 16;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hipFloatComplex m = hipCmulf((in[i + d]), hipConjf(in[i]));

        float cplx_mag = in[i].x * in[i].x + in[i].y * in[i].y;
        mag[i] = cplx_mag;
        out[i] = m;
    }
}


__global__ void mov_avg_cplx_kernel(
    hipFloatComplex* in, float* mag, hipFloatComplex* out, float* cor, int n)
{
    // int d = 16;
    int w = 48;
    int w2 = 64;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {

        hipFloatComplex sum = make_hipFloatComplex(0, 0);
        for (int j = 0; j < w; j++) {
            sum.x += in[i + j].x;
            sum.y += in[i + j].y;
        }

        float fsum = 0;
        for (int j = 0; j < w2; j++) {
            fsum += mag[i + j];
        }

        // __syncthreads();
        // if (i < n-63-16) {
        out[i] = sum;
        // }
        if (fsum == 0)
            cor[i] = 0;
        else
            cor[i] = sqrt(sum.x * sum.x + sum.y * sum.y) / fsum;
    }
}


void exec_corr_abs(hipFloatComplex* in,
                   hipFloatComplex* out,
                   float* mag,
                   int n,
                   int grid_size,
                   int block_size,
                   hipStream_t stream)
{
    corr_abs_kernel<<<grid_size, block_size, 0, stream>>>(in, out, mag, n + 63);
}

void exec_mov_avg(hipFloatComplex* in,
                  float* mag,
                  hipFloatComplex* out,
                  float* cor,
                  int n,
                  int grid_size,
                  int block_size,
                  hipStream_t stream)
{
    mov_avg_cplx_kernel<<<grid_size, block_size, 0, stream>>>(in, mag, out, cor, n);
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, mov_avg_cplx_kernel, 0, 0);
}

}
} // namespace wifi
} // namespace gr